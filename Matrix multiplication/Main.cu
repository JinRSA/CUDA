#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include "Matrix.h"
#define COUNT 10
#define HANDLE_ERROR(hipError_t) HandleError(hipError_t, __FILE__, __LINE__)

static void HandleError(const hipError_t err, const char* const in_file, const int line)
{
	if (err != hipSuccess)
	{
		printf("%s,\n� ����� %s,\n� ������: %d.\n", hipGetErrorString(err), in_file, line);
		exit(EXIT_FAILURE);
	}
}
//#define __global__

// [r1 x c1] * [r2 x c2] = [r1 x c2].
// c1 == r2!
__global__ void MatrixMultiplication(const el_t* const in_A, const el_t* const in_B, el_t* const out_Res, const uint rowsA, const uint colsA, const uint colsB)
{
	if (blockIdx.y < rowsA && blockIdx.x < colsB)
	{
		el_t summ = 0;
		for (uint i = 0; i < colsA; ++i)
			summ += in_A[colsA * blockIdx.y + i] * in_B[colsB * i + blockIdx.x];
		out_Res[colsB * blockIdx.y + blockIdx.x] = summ;
	}
}

int main()
{
	using namespace std;
	const int q = 8192, n = 4096, m = 6144;
	Matrix A = Init(n, q);	// �������������� ������� A.
	for (el_t i = 0; i < n * q; ++i)
		A.El[i] = i + 1;
	Matrix B = Init(q, m);	// �������������� ������� B.
	for (el_t i = 0; i < q * m; ++i)
		B.El[i] = i + 1;
	Matrix MatResCPU = InitZeros(A.Row, B.Col);	// �������������� ������ ������� ��������� ��������� �� CPU.
	Matrix MatResGPU = InitZeros(A.Row, B.Col);	// �������������� ������ ������� ��������� ��������� �� GPU.
	el_t* dev_A, *dev_B, *dev_MatRes;
	// �������� ������ �� GPU ��� ������.
	HANDLE_ERROR(hipMalloc((void**)&dev_A, A.Row * A.Col * sizeof(el_t)));
	HANDLE_ERROR(hipMalloc((void**)&dev_B, B.Row * B.Col * sizeof(el_t)));
	HANDLE_ERROR(hipMalloc((void**)&dev_MatRes, MatResGPU.Row * MatResGPU.Col * sizeof(el_t)));
	// ����������� ������� �� GPU.
	HANDLE_ERROR(hipMemcpy(dev_A, A.El, A.Row * A.Col * sizeof(el_t), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_B, B.El, B.Row * B.Col * sizeof(el_t), hipMemcpyHostToDevice));
	double TimesMultiplyByRow[COUNT], TimesMatrixMultiplication[COUNT];	// ������� ��� �������� ��������� ������ �� ���������� ���������.
	for (int i = 0; i < COUNT; ++i)	// �������� ����� ���������� ����� ���������� COUNT ���.
	{
		FillZeros(&MatResCPU);
		clock_t Start = clock();
		MultiplyByRow(&A, &B, &MatResCPU);	// ��������� ��������� ������ �� CPU.
		clock_t End = clock();
		TimesMultiplyByRow[i] = ((double)End - Start) / CLOCKS_PER_SEC;
		Start = clock();
		MatrixMultiplication <<<dim3(m, n), 1 >>>(dev_A, dev_B, dev_MatRes, A.Row, A.Col, B.Col);	// ��������� ��������� ������ �� GPU.
		End = clock();
		TimesMatrixMultiplication[i] = ((double)End - Start) / CLOCKS_PER_SEC;
	}
	HANDLE_ERROR(hipMemcpy(MatResGPU.El, dev_MatRes, MatResGPU.Row * MatResGPU.Col * sizeof(el_t), hipMemcpyDeviceToHost));	// ����������� ��������� � ���.
	// ���������� GPU ������.
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_MatRes);
	free(A.El);
	free(B.El);
	double Sum = 0;
	for (int i = 0; i < COUNT; ++i)
	{
		printf("��������� �� CPU %i:\t%f:\n", i + 1, TimesMultiplyByRow[i]);
		Sum += TimesMultiplyByRow[i];
	}
	printf("��������� �� CPU (avr.):\t%f:\n", Sum / COUNT);
	Sum = 0;
	for (int i = 0; i < COUNT; ++i)
	{
		printf("��������� �� GPU %i:\t%f:\n", i + 1, TimesMatrixMultiplication[i]);
		Sum += TimesMatrixMultiplication[i];
	}
	printf("��������� �� GPU (avr.):\t%f:\n", Sum / COUNT);
	const auto exitCode = !VerifyMatrix(&MatResCPU, &MatResGPU);
	free(MatResCPU.El);
	free(MatResGPU.El);
	return exitCode;
}

//nvcc Main.cu Matrix.cpp - O3